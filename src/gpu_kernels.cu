#include "hip/hip_runtime.h"
#pragma once

#include <glm/glm.hpp>
#include <glm/vec3.hpp>
#include <vector>
#include <hip/hip_runtime.h>
#include <cstdio>
using namespace std;

namespace gpu {
	
	namespace SingleGMM {

        float calculateMahalanobisDistance(glm::vec3 a,glm::vec3 b,mat3 covar) {
	        glm::mat3 covarInv= glm::inverse(covar);
            glm::vec3 temp = (a-b) * covarInv;
	        float distance = glm::outerProduct(a-b,temp);
            return distance;
        }

        float calculatePrior(glm::vec3 mean,glm::mat3 covar, glm::vec3 point) {
            return exp(-0.5*calculateMahalanobisDistance(point,mean,covar));
        }

        _global__ void calculateExpectation(int m,int N,float *priorProb, glm::vec3 *points, glm::vec3 *mean, glm::mat3 *covar) {
	        int index = (blockIdx.x * blockDim.x) + threadIdx.x;
	        if (index >= N)
		        return;

	        float sum = 0;
	        for (int i = 0; i < m; i++) {
                priorProb[i] = calculatePrior(mean[i],covar[i],points[index]);
		        sum += priorProb[i];
	        }

            for (int i = 0; i < m; i++) {
                priorProb[i] = priorProb[i]/sum;
            }

        }

        __global__ void calculateMaximization(int m,float *priorProb, glm::vec3 *points, glm::vec3 mean, glm::mat3 *covar) {
            int index = (blockIdx.x * blockDim.x) + threadIdx.x;
	        if (index >= N)
		        return;

        }

	}
	
    namespace HierGMM {

    }
	
}
