#pragma once

#include <glm/glm.hpp>
#include <glm/vec3.hpp>
#include <glm/mat3.hpp>
#include <vector>
#include <hip/hip_runtime.h>
#include <cstdio>
#include<hip/hip_runtime.h>
#include "gmm_kernels.h"

#define checkCUDAErrorWithLine(msg) checkCUDAError(msg, __LINE__)

/**
* Check for CUDA errors; print and exit if there was a problem.
*/
void checkCUDAError(const char *msg, int line = -1) {
	hipError_t err = hipGetLastError();
	if (hipSuccess != err) {
		if (line >= 0) {
			fprintf(stderr, "Line %d: ", line);
		}
		fprintf(stderr, "Cuda error: %s: %s.\n", msg, hipGetErrorString(err));
		exit(EXIT_FAILURE);
	}
}

/*****************
* Configuration *
*****************/

/*! Block size used for CUDA kernel launch. */
#define blockSize 128

/*! Size of the starting area in simulation space. */
#define scene_scale 0.1f


/***********************************************
* Kernel state (pointers are device pointers) *
***********************************************/

int numObjects;
dim3 threadsPerBlock(blockSize);

// Buffers allocated for the logic
glm::vec3 *dev_points;


void expectationStep(glm::vec3 *data,glm::vec3 *mean, glm::mat3 *covar, float *priors, float *prob, int N) {
	
	int numObjects = N;
	dim3 fullBlocksPerGrid((numObjects + blockSize - 1) / blockSize);

	hipMalloc((void**)&points, numObjects * sizeof(glm::vec3));
	checkCUDAErrorWithLine("hipMalloc dev_points failed!");

}

void maximizationStep(glm::vec3 *data, glm::vec3 *mean, glm::mat3 *covar, float *priors, float *prob, int N) {

	int numObjects = N;
	dim3 fullBlocksPerGrid((numObjects + blockSize - 1) / blockSize);

	hipMalloc((void**)&points, numObjects * sizeof(glm::vec3));
	checkCUDAErrorWithLine("hipMalloc dev_points failed!");

}

void expectationStep(glm::vec2 *data, glm::vec2 *mean, glm::mat2 *covar, float *priors, float *prob, int N) {

	int numObjects = N;
	dim3 fullBlocksPerGrid((numObjects + blockSize - 1) / blockSize);

	hipMalloc((void**)&points, numObjects * sizeof(glm::vec2));
	checkCUDAErrorWithLine("hipMalloc dev_points failed!");

}

void maximizationStep(glm::vec2 *data, glm::vec2 *mean, glm::mat2 *covar, float *priors, float *prob, int N) {

	int numObjects = N;
	dim3 fullBlocksPerGrid((numObjects + blockSize - 1) / blockSize);

	hipMalloc((void**)&points, numObjects * sizeof(glm::vec2));
	checkCUDAErrorWithLine("hipMalloc dev_points failed!");

}